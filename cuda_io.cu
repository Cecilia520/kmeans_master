#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>     //strtok(): breaking a string into a series of tokens
#include <sys/types.h>  /* open() */
#include <sys/stat.h>
#include <fcntl.h>
#include <unistd.h>     /* read(), close() */

#include "kmeans.h"
#define MAXNUM_PERLINE 128

////////////////////////////////////////////////////////////////
/*Read in file into the critical format for cluster analysis*///
////////////////////////////////////////////////////////////////

float** file_read(char *filename, int *numObs, int *numvariables)
/*
 argument instruction
 filename: input file name
 numObs: number of data observations (local)
 numvariables: number of the data features (variables)
*/
{
    float **dat;
    int     i, j, len;

    FILE *infile;
    char *line, *ret;
	//char *line;
    int   lineLen;

    // open the data file to prepare for read in
    infile = fopen(filename, "r");

    // first find the number of objects
    lineLen = MAXNUM_PERLINE;
    line = (char*) malloc(lineLen);

    (*numObs) = 0;
    while (fgets(line, lineLen, infile) != NULL) {
        /* check each line to find the max line length */
        while (strlen(line) == lineLen-1) {
            /* this line read is not complete */
            len = strlen(line);
            fseek(infile, -len, SEEK_CUR);
            /* increase lineLen */
            lineLen += MAXNUM_PERLINE;
            line = (char*) realloc(line, lineLen);

            ret = fgets(line, lineLen, infile);
        }

        if (strtok(line, " \t\n") != 0) // check the number of observations when there is a "\n"
            (*numObs)++;
    }
    rewind(infile);
    printf("lineLen = %d\n",lineLen);

    // find the number of variables
    (*numvariables) = 0;
    while (fgets(line, lineLen, infile) != NULL) {
        if (strtok(line, " \t\n") != 0) {
            /* ignore the id (first coordiinate): numvariables = 1; */
            while (strtok(NULL, " ,\t\n") != NULL) (*numvariables)++;
            break; /* this makes read from 1st object */
        }
    }
    rewind(infile);

    printf("File (%s) has %d number of Observations.\n",filename,*numObs);
    printf("File (%s) has %d number of Variables.\n",filename,*numvariables);

    /* allocate space for objects[][] and read all objects */
    len = (*numObs) * (*numvariables);
    dat    = (float**)malloc((*numObs) * sizeof(float*));
    dat[0] = (float*) malloc(len * sizeof(float));
    for (i=1; i<(*numObs); i++)
        dat[i] = dat[i-1] + (*numvariables);

    i=0;
    /* read all objects */
    while (fgets(line, lineLen, infile) != NULL) {
        if (strtok(line, " \t\n") == NULL) continue;
        for (j=0; j<(*numvariables); j++)
            dat[i][j] = atof(strtok(NULL, " ,\t\n"));
        i++;
    }

    fclose(infile);
    free(line);

    return dat;
}

////////////////////////////////////////////////////////////////
///////*Write the Results from calculation to text file*////////
////////////////////////////////////////////////////////////////
int file_write(char *filename, int numClusters, int numObs, int numvariables, float **clusters, int *prediction)
/* dimension information in the behind*/
/* input file name */
/* no. clusters */
/* no. data objects */
/* no. variables (local) */
/* [numClusters][numvariables] centers */
/* [numObs] */
{
    FILE *fptr;
    int   i, j;
    char  outFileName[1024];

    /* output: the variables of the cluster centres ----------------------*/
    sprintf(outFileName, "%s.cluster_centres", filename);
    printf("Writing variables of K=%d cluster centers to file \"%s\"\n",
           numClusters, outFileName);
    fptr = fopen(outFileName, "w");
    for (i=0; i<numClusters; i++) {
        fprintf(fptr, "%d ", i);
        for (j=0; j<numvariables; j++)
            fprintf(fptr, "%f ", clusters[i][j]);
        fprintf(fptr, "\n");
    }
    fclose(fptr);

    /* output: the closest cluster centre to each of the data points --------*/
    sprintf(outFileName, "%s.prediction", filename);
    printf("Writing prediction of N=%d data objects to file \"%s\"\n",
           numObs, outFileName);
    fptr = fopen(outFileName, "w");
    for (i=0; i<numObs; i++)
        fprintf(fptr, "%d %d\n", i, prediction[i]);
    fclose(fptr);

    return 1;
}
