#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include "kmeans.h"

static inline void _safe_cuda_call(hipError_t err, const char* msg, const char* file_name, const int line_number){
	if(err!=hipSuccess){
		fprintf(stderr,"%s\n\nFile: %s\n\nLine Number: %d\n\nReason: %s\n",msg,file_name,line_number,hipGetErrorString(err));
		std::cin.get();
		exit(EXIT_FAILURE);
	}
}

#define SAFE_CALL(call,msg) _safe_cuda_call((call),(msg),__FILE__,__LINE__)

#define STREAMS_CNT 4

static inline int nextPowerOfTwo(int n) {
	n--;

	n = n >>  1 | n;
	n = n >>  2 | n;
	n = n >>  4 | n;
	n = n >>  8 | n;
	n = n >> 16 | n;
	//  n = n >> 32 | n;    //  For 64-bit ints

	return ++n;
}

/*----< euclid_dist_2() >----------------------------------------------------*/
/* square of Euclid distance between two multi-dimensional points            */
__host__ __device__ inline static
float euclid_dist_2(int    numvariables,
		int    numObjs,
		int    numClusters,
		float *objects,     // [numvariables][numObjs]
		float *clusters,    // [numvariables][numClusters]
		int    objectId,
		int    clusterId) {
	float ans=0.0;
	for (int i = 0; i < numvariables; i++) {
		ans +=(objects[numObjs*i+objectId] - clusters[numClusters * i + clusterId])*(objects[numObjs * i + objectId] - clusters[numClusters * i + clusterId]);
	}

	return(ans);
}

/*----< find_nearest_cluster() >---------------------------------------------*/
__global__ static void find_nearest_cluster(int numvariables,
		int numObjs,
		int numClusters,
		float *objects,           //  [numvariables][numObjs]
		float *deviceClusters,    //  [numvariables][numClusters]
		int *membership,          //  [numObjs]
		int *intermediates){

	//  The type chosen for membershipChanged must be large enough to support reductions!
	//  There are blockDim.x elements, one for each thread in the block.
	//  See numThreadsPerClusterBlock in cuda_kmeans().
	//unsigned char *membershipChanged = (unsigned char *)sharedMemory;

	__shared__ unsigned char membershipChanged[128];

	float *clusters = deviceClusters;
	membershipChanged[threadIdx.x] = 0;

	int objectId = blockDim.x * blockIdx.x + threadIdx.x;
	if (objectId < numObjs) {
		int   index, i;
		float dist, min_dist;

		/* find the cluster id that has min distance to object */
		index    = 0;
		min_dist = euclid_dist_2(numvariables, numObjs, numClusters, objects, clusters, objectId, 0);
		for (i=1; i<numClusters; i++) {
			dist = euclid_dist_2(numvariables, numObjs, numClusters, objects, clusters, objectId, i);
			/* no need square root */
			if (dist < min_dist) { /* find the min and its array index */
				min_dist = dist;
				index    = i;
			}
		}

		if (membership[objectId] != index) {
			membershipChanged[threadIdx.x] = 1;
		}

		/* assign the membership to object objectId */
		membership[objectId] = index;

		__syncthreads();    //  For membershipChanged[]

		// blockDim.x *must* be a power of two!
		// this is a reduction
		for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
			if (threadIdx.x < s) {
				membershipChanged[threadIdx.x] += membershipChanged[threadIdx.x + s];
			}
			__syncthreads();
		}

		if (threadIdx.x == 0) {
			intermediates[blockIdx.x] = membershipChanged[0];
		}
	}
}

__global__ static void compute_delta(int *deviceIntermediates,
		int numIntermediates,    //  The actual number of intermediates
		int numIntermediates2)   //  The next power of two
{
	//  The number of elements in this array should be equal to numIntermediates2, the number of threads launched.
	//  It *must* be a power of two!
	extern __shared__ unsigned int intermediates[];

	//  Copy global intermediate values into shared memory.
	intermediates[threadIdx.x] = (threadIdx.x < numIntermediates) ? deviceIntermediates[threadIdx.x] : 0;

	__syncthreads();

	//  numIntermediates2 *must* be a power of two!
	for (unsigned int s = numIntermediates2 / 2; s > 0; s >>= 1) {
		if (threadIdx.x < s) {
			intermediates[threadIdx.x] += intermediates[threadIdx.x + s];
		}
		__syncthreads();
	}

	if (threadIdx.x == 0) {
		deviceIntermediates[0] = intermediates[0];
	}
}

/*----< cuda_kmeans() >-------------------------------------------------------*/
//
//  ----------------------------------------
//  DATA LAYOUT
//
//  objects         [numObjs][numvariables]
//  clusters        [numClusters][numvariables]
//  dimObjects      [numvariables][numObjs]
//  dimClusters     [numvariables][numClusters]
//  newClusters     [numvariables][numClusters]
//  deviceObjects   [numvariables][numObjs]
//  deviceClusters  [numvariables][numClusters]
//  ----------------------------------------
//
/* return an array of cluster centers of size [numClusters][numvariables]       */
float** cuda_kmeans(float **objects,      /* in: [numObjs][numvariables] */
		int     numvariables,    /* no. features */
		int     numObjs,      /* no. objects */
		int     numClusters,  /* no. clusters */
		float   threshold,    /* % objects change membership */
		int    *membership,   /* out: [numObjs] */
		int    *loop_iterations)
{
	int      i, j, index, loop=0;
	int     *newClusterSize; /* [numClusters]: no. objects assigned in each
								new cluster */
	float    delta;          /* % of objects change their clusters */
	float	**dimObjects;
	float	**clusters;       /* out: [numClusters][numvariables] */
	float	**dimClusters;
	float	**newClusters;    /* [numvariables][numClusters] */

	float	*deviceObjects;
	float	*deviceClusters;
	int		*deviceMembership;
	int		*deviceIntermediates;

	//  Copy objects given in [numObjs][numvariables] layout to new
	//  [numvariables][numObjs] layout
	malloc2D(dimObjects, numvariables, numObjs, float);
	for (i = 0; i < numvariables; i++) {
		for (j = 0; j < numObjs; j++) {
			dimObjects[i][j] = objects[j][i];
		}
	}

	/* pick first numClusters elements of objects[] as initial cluster centers*/
	malloc2D(dimClusters, numvariables, numClusters, float);
	for (i = 0; i < numvariables; i++) {
		for (j = 0; j < numClusters; j++) {
			dimClusters[i][j] = dimObjects[i][j];
		}
	}

	/* initialize membership[] */
	for (i=0; i<numObjs; i++)
		membership[i] = -1;

	/* need to initialize newClusterSize and newClusters[0] to all 0 */
	newClusterSize = (int*)calloc(numClusters, sizeof(int));
	assert(newClusterSize != NULL);

	malloc2D(newClusters, numvariables, numClusters, float);
	memset(newClusters[0], 0, numvariables * numClusters * sizeof(float));

	// To support reduction, numThreadsPerClusterBlock *must* be a power of two,
	// and it *must* be no larger than the number of bits that will fit into an unsigned char,
	// the type used to keep track of membership changes in the kernel.
	const unsigned int numThreadsPerClusterBlock = 128;
	const unsigned int numClusterBlocks = (numObjs + numThreadsPerClusterBlock - 1) / numThreadsPerClusterBlock;
	//const unsigned int clusterBlockSharedDataSize = numThreadsPerClusterBlock * sizeof(unsigned char);
	const unsigned int numReductionThreads = nextPowerOfTwo(numClusterBlocks);
	const unsigned int reductionBlockSharedDataSize = numReductionThreads * sizeof(unsigned int);

	SAFE_CALL((hipMalloc(&deviceObjects, numObjs*numvariables*sizeof(float))), "CUDA malloc error!");
	SAFE_CALL((hipMalloc(&deviceClusters, numClusters*numvariables*sizeof(float))), "CUDA malloc error!");
	SAFE_CALL((hipMalloc(&deviceMembership, numObjs*sizeof(int))), "CUDA malloc error!");
	SAFE_CALL((hipMalloc(&deviceIntermediates, numReductionThreads*sizeof(unsigned int))), "CUDA malloc error!");

	SAFE_CALL(hipMemcpy(deviceObjects, dimObjects[0], numObjs*numvariables*sizeof(float), hipMemcpyHostToDevice), "CUDA memory release error!");
	SAFE_CALL(hipMemcpy(deviceMembership, membership, numObjs*sizeof(int), hipMemcpyHostToDevice), "CUDA memory release error!");

	do{
		SAFE_CALL((hipMemcpy(deviceClusters, dimClusters[0], numClusters*numvariables*sizeof(float), hipMemcpyHostToDevice)),
				"CUDA memory copy from host to device error!");

		find_nearest_cluster<<<numClusterBlocks, numThreadsPerClusterBlock>>>(numvariables,
				numObjs,
				numClusters,
				deviceObjects,
				deviceClusters,
				deviceMembership,
				deviceIntermediates);


		hipDeviceSynchronize();

		compute_delta<<<1,numReductionThreads, reductionBlockSharedDataSize >>>(deviceIntermediates, numClusterBlocks, numReductionThreads);
		hipDeviceSynchronize();

		int d;
		SAFE_CALL((hipMemcpy(&d, deviceIntermediates, sizeof(int), hipMemcpyDeviceToHost)), "CUDA memory copy from device to host error!");
		delta = (float)d;

		SAFE_CALL((hipMemcpy(membership, deviceMembership, numObjs*sizeof(int), hipMemcpyDeviceToHost)), "CUDA memory copy from device to host error!");

		for (i=0; i<numObjs; i++) {
			/* find the array index of nestest cluster center */
			index = membership[i];

			/* update new cluster centers : sum of objects located within */
			newClusterSize[index]++;
			for (j=0; j<numvariables; j++)
				newClusters[j][index] += objects[i][j];
		}

		//  TODO: Flip the nesting order
		//  TODO: Change layout of newClusters to [numClusters][numvariables]
		// average the sum and replace old cluster centers with newClusters
		for (i=0; i<numClusters; i++) {
			for (j=0; j<numvariables; j++) {
				if (newClusterSize[i] > 0)
					dimClusters[j][i] = newClusters[j][i] / newClusterSize[i];
				newClusters[j][i] = 0.0;   /* set back to 0 */
			}
			newClusterSize[i] = 0;   /* set back to 0 */
		}
		delta /= numObjs;
	} while (delta > threshold && loop++ < 500);

	*loop_iterations = loop + 1;

	/* allocate a 2D space for returning variable clusters[] (coordinates of cluster centers) */
	malloc2D(clusters, numClusters, numvariables, float);
	for (i = 0; i < numClusters; i++) {
		for (j = 0; j < numvariables; j++) {
			clusters[i][j] = dimClusters[j][i];
		}
	}

	SAFE_CALL((hipFree(deviceObjects)), "CUDA memory release error!");
	SAFE_CALL((hipFree(deviceClusters)), "CUDA memory release error!");
	SAFE_CALL((hipFree(deviceMembership)),"CUDA memory release error!");
	SAFE_CALL((hipFree(deviceIntermediates)),"CUDA memory release error!");

	free(dimObjects[0]);
	free(dimObjects);
	free(dimClusters[0]);
	free(dimClusters);
	free(newClusters[0]);
	free(newClusters);
	free(newClusterSize);

	return clusters;
}
